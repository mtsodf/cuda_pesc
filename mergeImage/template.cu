#include "hip/hip_runtime.h"
//-----------------------------------------
// Autor: Farias
// Data : January 2012
// Goal : Image treatment
//-----------------------------------------

/***************************************************************************************************
	Includes
***************************************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

#include "rf-time.h"


/***************************************************************************************************
	Defines
***************************************************************************************************/

#define ELEM(i,j,DIMX_) (i+(j)*(DIMX_))


/***************************************************************************************************
	Functions
***************************************************************************************************/

using namespace std;


/**************************************************************************************************/

__host__ void erro( const char tipoDeErro[] ) {

	fprintf( stderr, "%s\n", tipoDeErro );
	exit(0);

}


/**************************************************************************************************/
__host__ void savePPM( char *fname, unsigned char *buffer, int width, int height ) {

	if( !buffer ) {
		cout << "Image not saved. This ViewPoint Class in NOT FULL." << endl;
		return;
	}
	FILE *f = fopen( fname, "wb" );
	if( f == NULL ) erro( "Error writting PPM file." );

	fprintf( f, "P6\n# Written in the CUDA course\n%u %u\n%u\n", width, height, 255 );
	fwrite( buffer, 3, width*height, f );
	fclose(f);

}

__host__ void readPPM( char *fname, unsigned char **buffer, int *width, int *height ) {

	char aux[256];
	FILE *f = fopen( fname, "rb" );
	if( f == NULL ) 
		erro( "Error reading PPM image" );

	fgets( aux, 256, f );
	fgets( aux, 256, f );
	fgets( aux, 256, f );
	sscanf( aux, "%d %d", width, height );
	fgets( aux, 256, f );

	if( *buffer ) {
		free( *buffer );
	}
	
	int size = 3*(*width)*(*height)*sizeof( unsigned char );
	cout << "Image dimension: (" << *width << "," << *height <<")\n";

	if( ( *buffer = (unsigned char *)malloc( size ) ) == NULL )
		erro( "Error allocating image" );

	fread( *buffer, 3, (*width)*(*height), f );
	fclose( f );

}

/**************************************************************************************************/
__global__ void funcGPU( int width, int height, unsigned char *src, unsigned char *dest ) {

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;

	if( i < width && j < height ) {

		int idx = 3*ELEM( i, j, width );
		int r = src[ idx+2 ];
		int g = src[ idx+1 ];
		int b = src[ idx   ];
		dest[ idx   ] = (unsigned char)b;
		dest[ idx+1 ] = (unsigned char)g;
		dest[ idx+2 ] = (unsigned char)r;

	}

}

/**************************************************************************************************/
__global__ void mergeImages( int width, int height, unsigned char *src1, unsigned char *src2, unsigned char *dest ) {

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;

	if( i < width && j < height ) {

		int idx = 3*ELEM( i, j, width );
		int r1 = src1[ idx+2 ]; int r2 = src2[ idx+2 ];
		int g1 = src1[ idx+1 ]; int g2 = src2[ idx+1 ];
		int b1 = src1[ idx   ]; int b2 = src2[ idx   ];
	
		dest[ idx   ] = (unsigned char)(0.5*b1 + 0.5*b2);
		dest[ idx+1 ] = (unsigned char)(0.5*g1 + 0.5*g2);
		dest[ idx+2 ] = (unsigned char)(0.5*r1 + 0.5*r2);

	}

}


/**************************************************************************************************/
__host__ int main( int argc, char *argv[] ) {

	double start_time, gpu_time;
	int    h_width, h_height;
	unsigned char*  h_image_1 = NULL, *h_image_2 = NULL, *h_res = NULL;
	
	if( argc != 3 ) {
		
		erro( "Sintaxe: template image" );
		
	}


        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        // This function call returns 0 if there are no CUDA capable devices.
        if( deviceCount == 0 ) {
                printf("There is no device supporting CUDA\n");
                exit( 1 );
        }

        if(deviceCount < 2){
                printf("Nao tem placa grafica disponivel\n");
        }

        printf("Device Count %d\n", deviceCount);

        hipSetDevice(1);


	cout << "Program for image treatment " << endl;

	readPPM( argv[1], &h_image_1, &h_width, &h_height );
	readPPM( argv[2], &h_image_2, &h_width, &h_height );

	int size = 3*h_width*h_height*sizeof( unsigned char );

	// Buffer for result image
	if( ( h_res = (unsigned char *)malloc( size ) ) == NULL )
		erro( "Erro allocating result imagem buffer." );

	// Allocate memory for buffers in the GPU
	unsigned char *d_image_1;
	hipMalloc( (void**)&d_image_1, size );
	hipMemcpy( d_image_1, h_image_1, size, hipMemcpyHostToDevice );
	unsigned char *d_res;
	hipMalloc( (void**)&d_res, size );

	// Allocate memory for buffers in the GPU
	unsigned char *d_image_2;
	hipMalloc( (void**)&d_image_2, size );
	hipMemcpy( d_image_2, h_image_2, size, hipMemcpyHostToDevice );


	// Calcula dimensoes da grid e dos blocos
	dim3 blockSize( 16, 16 );

	int numBlocosX = h_width  / blockSize.x + ( h_width  % blockSize.x == 0 ? 0 : 1 );
	int numBlocosY = h_height / blockSize.y + ( h_height % blockSize.y == 0 ? 0 : 1 );
	dim3 gridSize( numBlocosX, numBlocosY, 1 );

	cout << "Blocks (" << blockSize.x << "," << blockSize.y << ")\n";
	cout << "Grid   (" << gridSize.x << "," << gridSize.y << ")\n";

	start_time = get_clock_msec();
	mergeImages<<< gridSize, blockSize >>>( h_width, h_height, d_image_1, d_image_2, d_res );
	hipDeviceSynchronize();
	gpu_time = get_clock_msec() - start_time;

	// Copy result buffer back to cpu memory
	hipMemcpy( h_res, d_res, size, hipMemcpyDeviceToHost );

	// Salva imagem resultado
	savePPM( (char *)"template.ppm", h_res, h_width, h_height );
	
	// Imprime tempo
	cout << "\tTempo de execucao da GPU: " << gpu_time << endl;
	cout << "-------------------------------------------" << endl;

	// Free buffers
	hipFree( d_image_1 );
	hipFree( d_image_2 );
	hipFree( d_res   );
	free( h_image_1 );
	free( h_image_2 );
	free( h_res );

	//system( "eog template.ppm" );

	return 0;

}
