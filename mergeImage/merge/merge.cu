#include "hip/hip_runtime.h"
//-----------------------------------------
// Autor: Farias
// Data : May 2011
// Goal : Merge duas imagens PPM
//-----------------------------------------

/***************************************************************************************************
	Includes
***************************************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

#include "rf-time.h"


/***************************************************************************************************
	Defines
***************************************************************************************************/

#define ELEM(i,j,DIMX_) ((i)+(j)*(DIMX_))
#define STREAM_SIZE 256

/***************************************************************************************************
	Functions
***************************************************************************************************/

using namespace std;


/**************************************************************************************************/

__host__ void erro( const char tipoDeErro[] ) {

	fprintf( stderr, "%s\n", tipoDeErro );
	exit(0);

}


/**************************************************************************************************/
__host__ void salvaPPM( char *fname, unsigned char *buffer, int width, int height ) {

	if( !buffer ) {
		cout << "Image not saved. This ViewPoint Class in NOT FULL." << endl;
		return;
	}
	FILE *f = fopen( fname, "wb" );
	if( f == NULL ) erro( "Erro escrevendo o PPM" );

	fprintf( f, "P6\n# Gravado no curso de CUDA\n%u %u\n%u\n", width, height, 255 );
	fwrite( buffer, 3, width*height, f );
	fclose(f);

}

__host__ void lerPPM( char *fname, unsigned char **buffer, int *width, int *height ) {

	char aux[256];
	FILE *f = fopen( fname, "r" );
	if( f == NULL ) 
		erro( "Erro lendo o PPM" );

	fgets( aux, 256, f );
	fgets( aux, 256, f );
	fgets( aux, 256, f );
	sscanf( aux, "%d %d", width, height );
	fgets( aux, 256, f );

	if( !(*buffer) ) {
		free( *buffer );
	}
	
	int size = 3*(*width)*(*height)*sizeof( char );
	cout << "Dimensao da imagem: (" << *width << "," << *height <<")\n";

	if( ( *buffer = (unsigned char *)malloc( size ) ) == NULL )
		erro( "Erro lendo o buffer da imagem" );

	fread( *buffer, 3, (*width)*(*height), f );
	fclose( f );

}

/**************************************************************************************************/
__global__ void mergeGPU( unsigned char *image1, unsigned char *image2, 
			  unsigned char *res, int width, int height ) {

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;

	if( i < width && j < height ) {

	 	int idx = 3*ELEM( i, j, width );
	 	int r1 = image1[ idx+2 ];
	 	int g1 = image1[ idx+1 ];
	 	int b1 = image1[ idx   ];
	 	int r2 = image2[ idx+2 ];
	 	int g2 = image2[ idx+1 ];
	 	int b2 = image2[ idx   ];
	 	int r = (int)( ( (float)r1 + (float)r2 )*0.5f );
		int g = (int)( ( (float)g1 + (float)g2 )*0.5f );
	 	int b = (int)( ( (float)b1 + (float)b2 )*0.5f );
	 	res[ idx+2 ] = (unsigned char)r;
	 	res[ idx+1 ] = (unsigned char)g;
	 	res[ idx   ] = (unsigned char)b;
		
	 }
	
}


/**************************************************************************************************/
__host__ int main( int argc, char *argv[] ) {

	int blSizeX = 16, blSizeY = 16;
	double start_time, gpu_time;
	int    h_width1, h_height1;
	int    h_width2, h_height2;
	unsigned char   *h_imagem1 = NULL, *h_imagem2 = NULL;
	unsigned char   *h_imagem_resultado = NULL;
	
	if( argc < 3 ) {
		
		erro( "Syntaxe: merge fig1 fig2 [numBlocoX numBlocoY]" );
		
	}


        hipSetDevice(1);


	cout << "Programa para Merge duas Imagens PPM " << endl;

	switch( argc ) {

	case 4:
		blSizeX = blSizeY = atoi( argv[ 3 ] );
		break;
	case 5:
		blSizeX = atoi( argv[ 3 ] );
		blSizeY = atoi( argv[ 4 ] );
	}

	lerPPM( argv[1], &h_imagem1, &h_width1, &h_height1 );
	lerPPM( argv[2], &h_imagem2, &h_width2, &h_height2 );

	if( h_width1 != h_width2 || h_height1 != h_height2 )
		erro( "Imagens tem dimensoes diferentes.\nAbortando." );

	int size = 3*h_width1*h_height1*sizeof( char );

	if( ( h_imagem_resultado = (unsigned char *)malloc( size ) ) == NULL )
		erro( "Erro alocando imagem resultado." );

	// Aloca memória no device e copia vetorA e vetorB para lá
	unsigned char *d_imagem1 = NULL;
	hipMalloc( (void**)&d_imagem1, size );
	hipMemcpy( d_imagem1, h_imagem1, size, hipMemcpyHostToDevice );

	unsigned char *d_imagem2 = NULL;
	hipMalloc( (void**)&d_imagem2, size );
	hipMemcpy( d_imagem2, h_imagem2, size, hipMemcpyHostToDevice );

	unsigned char *d_res = NULL;
	hipMalloc( (void**)&d_res, size );

	// Calcula dimensoes da grid e dos blocos
	dim3 blockSize( blSizeX, blSizeY );
	int numBlocosX = h_width1  / blockSize.x + ( h_width1  % blockSize.x == 0 ? 0 : 1 );
	int numBlocosY = h_height1 / blockSize.y + ( h_height1 % blockSize.y == 0 ? 0 : 1 );
	dim3 gridSize( numBlocosX, numBlocosY, 1 );

	cout << "Blocks (" << blockSize.x << "," << blockSize.y << ")\n";
	cout << "Grid   (" << gridSize.x << "," << gridSize.y << ")\n";

	// Chama SomarVetoresGPU
	start_time = get_clock_msec();
	mergeGPU<<< gridSize, blockSize >>>( d_imagem1, d_imagem2, d_res, h_width1, h_height1 );
	hipDeviceSynchronize();
	gpu_time = get_clock_msec() - start_time;

	// Copia o resultado de volta para o host
	hipMemcpy( h_imagem_resultado, d_res, size, hipMemcpyDeviceToHost );

	// Salva imagem resultado
	salvaPPM( "merge.ppm", h_imagem_resultado, h_width1, h_height1 );
	
	// Imprime tempo
	cout << "\tTempo de execucao da GPU: " << gpu_time << "ms" << endl;
	cout << "-------------------------------------------" << endl;

	system( "eog merge.ppm" );	

	// Libera memória do device
	hipFree( d_imagem1 );
	hipFree( d_imagem2 );
	hipFree( d_res     );

	//Merge das imagens com stream
	hipStream_t    stream0;

	// initialize the streams
	hipStreamCreate( &stream0 )	
	
	unsigned char   *h_imagem1_pin = NULL, *h_imagem2_pin = NULL;
	unsigned char   *h_imagem_resultado = NULL;	
	
	// Alocando o valor de 
	hipHostAlloc( (void**)&h_imagem1_pin, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault );
	hipHostAlloc( (void**)&h_imagem2_pin, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault );

	for(int i = 0; i < size; i++){

		h_imagem1_pin[i] = h_imagem1[i];
		h_imagem2_pin[i] = h_imagem2[i];

	}

	int pixels = size/3;

	


	for(int i = 0; i < pixels; i += STREAM_SIZE){
		
	}
	



	hipHostFree( h_imagem1_pin );
	hipHostFree( h_imagem2_pin );
	
	// Libera memória do host
	free( h_imagem1 );
	free( h_imagem2 );
	free( h_imagem_resultado );

	return 0;

}
